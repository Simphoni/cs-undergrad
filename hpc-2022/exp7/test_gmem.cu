#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>

// You should modify this parameter.
#define STRIDE 5

__global__ void stride_copy(float *dst, float *src) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    dst[i * STRIDE] = src[i * STRIDE];
}

int main() {
    float *dev_a = 0, *dev_b = 0;
    int size = (1 << 24);
    hipMalloc((void **)&dev_a, size  * 32 * sizeof(float));
    hipMalloc((void **)&dev_b, size  * 32 * sizeof(float));
    dim3 gridSize(size / 1024, 1);
    dim3 blockSize(1024, 1);

    hipEvent_t st, ed;
    hipEventCreate(&st);
    hipEventCreate(&ed);
    float duration;

    // The parameters that you should change.
    for (int t = 0; t < 1024; t++) {
        stride_copy<<<gridSize, blockSize>>>(dev_b, dev_a);
    }
    hipEventRecord(st, 0);
    for (int t = 0; t < 1024; t++) {
        stride_copy<<<gridSize, blockSize>>>(dev_b, dev_a);
    }
    hipEventRecord(ed, 0);
    hipEventSynchronize(st);
    hipEventSynchronize(ed);
    hipEventElapsedTime(&duration, st, ed);
    duration /= 1024;
    std::cout << "stride:    " << STRIDE << std::endl;
    std::cout << "bandwidth: " << 8 * size / duration / 1e6 << std::endl;
}

