#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <iostream>

// You should modify these parameters.
#define BITWIDTH 2
#define STRIDE 4

constexpr int times = 4096;

__global__ void test_shmem() {
#if (BITWIDTH == 2)
    volatile __shared__ uint16_t shm[32 * 128];
    volatile uint16_t tmp;
#elif (BITWIDTH == 4)
    volatile __shared__ uint32_t shm[32 * 128];
    volatile uint32_t tmp;
#elif (BITWIDTH == 8)
    volatile __shared__ uint64_t shm[32 * 128];
    volatile uint64_t tmp;
#endif

    for (int i = 0; i < times; i++) {
        tmp = shm[threadIdx.x * STRIDE];
    }
}

int main() {
    int size = (1 << 16);
    dim3 gridSize(size / 128, 1);
    dim3 blockSize(128, 1);

    hipEvent_t st, ed;
    hipEventCreate(&st);
    hipEventCreate(&ed);
    float duration;

    // Warm up.
    for (int t = 0; t < 1024; t++) {
        test_shmem<<<gridSize, blockSize>>>();
        cudaCheckError();
    }

    hipEventRecord(st, 0);
    for (int t = 0; t < 1024; t++) {
        test_shmem<<<gridSize, blockSize>>>();
        cudaCheckError();
    }
    hipEventRecord(ed, 0);
    hipEventSynchronize(st);
    hipEventSynchronize(ed);
    hipEventElapsedTime(&duration, st, ed);
    duration /= float(1024) * float(times);
    
    std::cout << "bitwidth:  " << BITWIDTH << std::endl;
    std::cout << "stride:    " << STRIDE << std::endl;
    std::cout << "bandwidth: " << size * BITWIDTH / duration / 1e6 << std::endl;
}

