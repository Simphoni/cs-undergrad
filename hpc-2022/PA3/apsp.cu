#include "hip/hip_runtime.h"
#include "apsp.h"
#include "cuda_utils.h"
#define BLOCK 64
#define INF 0x3f3f3f3f
#define TILE 16
#define NUM 4

namespace {
#define IDX(i, j, n) ((i) * (n) + (j))
  __global__ void raw_floyd(int n, int core_start, int *graph, int *core_core) {
    const int x = threadIdx.x;
    const int yi = threadIdx.y;
    const int core_stride = min(n - core_start, BLOCK);
    
    __shared__ int core[BLOCK][BLOCK + 1];
    
    int y = yi;
    for (int iter = 0; iter < TILE; iter ++) {
      // [core][core]
      if (y < core_stride && x < core_stride)
        core[y][x] = graph[IDX(core_start + y, core_start + x, n)];
      else core[y][x] = INF;
      y += NUM;
    }
    
    __syncthreads();
    for (int k = 0; k < BLOCK; k ++) {
      for (int i = yi; i < BLOCK; i += NUM) {
        int tmp = core[x][k] + core[k][i];
        if (core[x][i] > tmp) core[x][i] = tmp;
      }
      __syncthreads();
    }
    
    y = yi;
    for (int iter = 0; iter < TILE; iter ++) {
      // store back in graph[]
      if (y < core_stride && x < core_stride)
        graph[IDX(core_start + y, core_start + x, n)] = core[y][x];
      // store in a quick cache
      core_core[IDX(y, x, BLOCK)] = core[y][x];
      y += NUM;
    }
  }
  
  __global__ void expand_row(int n, int core_start, int *graph, int *core_core, int *core_dest) {
    // all that goes [core] -> C[core]
    // not the bottleneck
    const int x = threadIdx.x;
    const int yi = threadIdx.y;
    const int dest_start = blockIdx.x * BLOCK;
    if (dest_start == core_start) return;
    
    const int dest_stride = min(n - dest_start, BLOCK);
    const int core_stride = min(n - core_start, BLOCK);
    
    __shared__ int dest[BLOCK][BLOCK + 1], core[BLOCK][BLOCK + 1];
    
    for (int iter = 0, y = yi; iter < TILE; iter ++, y += NUM) {
      // [core][core]
      core[y][x] = core_core[IDX(y, x, BLOCK)];
    }
    for (int iter = 0, y = yi; iter < TILE; iter ++, y += NUM) {
      // [core][dest]
      if (y < core_stride && x < dest_stride)
        dest[y][x] = graph[IDX(core_start + y, dest_start + x, n)];
      else dest[y][x] = INF;
    }
    __syncthreads();
    
    for (int k = 0; k < BLOCK; k ++) {
      for (int i = yi; i < BLOCK; i += NUM) {
        int tmp = core[x][k] + dest[k][i];
        if (dest[x][i] > tmp) dest[x][i] = tmp;
      }
      __syncthreads();
    }

    for (int iter = 0, y = yi; iter < TILE; iter ++, y += NUM) {
      // [core][dest]
      if (y < core_stride && x < dest_stride)
        graph[IDX(core_start + y, dest_start + x, n)] = dest[y][x];
      core_dest[IDX(y + dest_start, x, BLOCK)] = dest[y][x];
    }
  }
  
  __global__ void expand_col(int n, int core_start, int *graph, int *core_core, int *src_core) {
    // all that goes into the core
    const int x = threadIdx.x;
    const int yi = threadIdx.y;
    const int src_start = blockIdx.x * BLOCK;
    if (src_start == core_start) return;
    
    const int src_stride = min(n - src_start, BLOCK);
    const int core_stride = min(n - core_start, BLOCK);

    __shared__ int core[BLOCK][BLOCK + 1], src[BLOCK][BLOCK + 1];
    
    for (int iter = 0, y = yi; iter < TILE; iter ++) {
      // [core][core]
      core[y][x] = core_core[IDX(y, x, BLOCK)];
      y += NUM;
    }
    for (int iter = 0, y = yi; iter < TILE; iter ++) {
      // [src][core]
      if (y < src_stride && x < core_stride)
        src[y][x] = graph[IDX(src_start + y, core_start + x, n)];
      else src[y][x] = INF;
      y += NUM;
    }
    __syncthreads();
    
    for (int k = 0; k < BLOCK; k ++) {
      for (int i = yi; i < BLOCK; i += NUM) {
        int tmp = src[x][k] + core[k][i];
        if (src[x][i] > tmp) src[x][i] = tmp;
      }
      __syncthreads();
    }
    
    for (int iter = 0, y = yi; iter < TILE; iter ++) {
      // [src][core]
      if (y < src_stride && x < core_stride)
        graph[IDX(src_start + y, core_start + x, n)] = src[y][x];
      y += NUM;
    }
    for (int iter = 0, y = yi; iter < TILE; iter ++) {
      // [src][core]
      src_core[IDX(src_start + y, x, BLOCK)] = src[y][x];
      y += NUM;
    }
  }

  __global__ void expand_all(int n, int core_start, int *graph, int *src_core, int *core_dest) {
    // src_start+i -> core_start+k -> dest_start+j
    const int src_start = blockIdx.x * BLOCK;
    const int dest_start = blockIdx.y * BLOCK;
    if (src_start == core_start || dest_start == core_start) return;
    const int i = threadIdx.x;
    const int ji = threadIdx.y;
    const int src_stride = min(n - src_start, BLOCK);
    const int core_stride = min(n - core_start, BLOCK);
    const int dest_stride = min(n - dest_start, BLOCK);

#define STEP 4
    int mysrc[STEP][NUM], mydest[STEP][NUM], myans[NUM][NUM];
    // please put these in register

    // const int de = i % 8 * 4 + (ji & 1) * 32;
    // const int sr = i / 8 * 4 + (ji & 2) * 16;
    const int de = threadIdx.x % 16 * 4;
    const int sr = (threadIdx.x / 16 * 4 + threadIdx.y) * 4;

    __shared__ int src[BLOCK][BLOCK], dest[BLOCK][BLOCK];

    for (int iter = 0, j= ji; iter < TILE; iter ++) {
      // [src][core]
      src[j][i] = src_core[IDX(src_start + j, i, BLOCK)];
      j += NUM;
    }

    for (int iter = 0, j = ji; iter < TILE; iter ++) {
      // [core][dest]
      dest[j][i] = core_dest[IDX(dest_start + j, i, BLOCK)];
      j += NUM;
    }
    __syncthreads();

    
    for (int p = 0; p < 4; p ++)
#pragma unroll
      for (int q = 0; q < 4; q ++)
        if (p + sr < src_stride && q + de < dest_stride)
          myans[p][q] = graph[IDX(src_start + sr + p, dest_start + de + q, n)];
    
    for (int iter = 0, w = 0; iter < BLOCK / STEP; iter ++, w += STEP) {
      for (int p = 0; p < STEP; p ++)
#pragma unroll
        for (int q = 0; q < 4; q ++)
          mysrc[p][q] = src[sr + q][w + p];
      
      for (int p = 0; p < STEP; p ++)
#pragma unroll
        for (int q = 0; q < 4; q ++)
          mydest[p][q] = dest[w + p][de + q];

#pragma unroll
      for (int p = 0; p < 4; p ++)
#pragma unroll
        for (int q = 0; q < 4; q ++)
#pragma unroll
          for (int k = 0; k < STEP; k ++)
            myans[p][q] = min(myans[p][q], mysrc[k][p] + mydest[k][q]);
    }
    for (int p = 0; p < 4; p ++)
#pragma unroll
      for (int q = 0; q < 4; q ++)
        if (p + sr < src_stride && q + de < dest_stride)
          graph[IDX(src_start + sr + p, dest_start + de + q, n)] = myans[p][q];
  }
}

#include <cstdio>
#include <unistd.h>
#include <sys/time.h>

namespace Timer {
  timeval start, cur;
  void kickoff() {
    gettimeofday(&start, NULL);
  }
  float timeEllapsed() {
    gettimeofday(&cur, NULL);
    return (cur.tv_usec - start.tv_usec) * 1e-6 + (cur.tv_sec - start.tv_sec);
  }
};

#define ROUNDUP(n) (((n) + BLOCK - 1) / BLOCK * BLOCK)

//#define DEBUG

void apsp(int n, /* device */ int *graph) {
#ifdef DEBUG
  double floyd = 0, ti = 0;
#endif
  int *core_core, *core_dest, *src_core;
  hipMalloc(&core_core, BLOCK * BLOCK * sizeof(int));
  hipMalloc(&src_core, BLOCK * ROUNDUP(n) * sizeof(int));
  hipMalloc(&core_dest, BLOCK * ROUNDUP(n) * sizeof(int));
  hipStream_t row, col;
  hipStreamCreate(&row);
  hipStreamCreate(&col);

  for (int i = 0; i < n; i += BLOCK) {
    dim3 thr( BLOCK, NUM );
    dim3 blk( 1, 1 );
    raw_floyd<<<blk, thr>>>(n, i, graph, core_core);
    hipDeviceSynchronize();

    blk = dim3( (n - 1) / BLOCK + 1, 1 );
#ifdef DEBUG
    Timer::kickoff();
#endif
    expand_row<<<blk, thr, 0, row>>>(n, i, graph, core_core, core_dest);
    expand_col<<<blk, thr, 0, col>>>(n, i, graph, core_core, src_core);
    hipDeviceSynchronize();
#ifdef DEBUG
    ti += Timer::timeEllapsed();
#endif

    blk = dim3( (n - 1) / BLOCK + 1,
                (n - 1) / BLOCK + 1 );
#ifdef DEBUG
    Timer::kickoff();
#endif
    expand_all<<<blk, thr>>>(n, i, graph, src_core, core_dest);
    hipDeviceSynchronize();
#ifdef DEBUG
    floyd += Timer::timeEllapsed();
#endif
  }
  hipFree(core_core);
  hipFree(core_dest);
  hipFree(src_core);
#ifdef DEBUG
  printf("%.6lfms\n", ti * 1000);
  printf("%.6lfms\n", floyd * 1000);
#endif
}

// 10000: 22715ms
