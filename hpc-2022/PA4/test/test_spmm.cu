#include "gtest/gtest.h"
#include "util.h"
#include "valid.h"
#include "spmm_ref.h"
#include "spmm_opt.h"
#include "spmm_cusparse.h"

class SpMMTest : public testing::Test
{
protected:
    vector<void *> tensor_ptr;
    float *p_in_feat_vec, *p_out_feat_vec, *p_out_feat_vec_ref, *p_value;
    CSR *g;
    virtual void SetUp()
    {
        p_in_feat_vec = allocate<float>(kNumV * kLen, &tensor_ptr);
        p_out_feat_vec = allocate<float>(kNumV * kLen, &tensor_ptr);
        p_out_feat_vec_ref = allocate<float>(kNumV * kLen, &tensor_ptr);
        p_value = allocate<float>(kNumE, &tensor_ptr);
        g = new CSR(kNumV, kNumE, gptr, gidx, p_value);
    }
    virtual void TearDown()
    {
        for (auto item : tensor_ptr)
        {
            hipFree(item);
        }
    }
};

TEST_F(SpMMTest, validation)
{
    SpMMRef *spmmer_ref = new SpMMRef(g, kLen);
    SpMMOpt *spmmer = new SpMMOpt(g, kLen);
    spmmer_ref->preprocess(p_in_feat_vec, p_out_feat_vec_ref);
    spmmer->preprocess(p_in_feat_vec, p_out_feat_vec);
    checkCudaErrors(hipMemset(p_out_feat_vec, 0, sizeof(float) * kNumV * kLen));
    checkCudaErrors(hipMemset(p_out_feat_vec_ref, 0, sizeof(float) * kNumV * kLen));
    spmmer_ref->run(p_in_feat_vec, p_out_feat_vec_ref);
    spmmer->run(p_in_feat_vec, p_out_feat_vec);
    checkCudaErrors(hipDeviceSynchronize());
    // This ASSERT will fail because your SpMM is not implemented yet
    ASSERT_LT(valid(p_out_feat_vec, p_out_feat_vec_ref, kNumV * kLen), kNumV * kLen / 10000 + 1);
}

TEST_F(SpMMTest, cusparse_performance)
{
    SpMMCuSparse *spmmer = new SpMMCuSparse(g, kLen);
    spmmer->preprocess(p_in_feat_vec, p_out_feat_vec);
    auto time = getAverageTimeWithWarmUp([&]()
                                         { spmmer->run(p_in_feat_vec, p_out_feat_vec); });
    dbg(time);
}

TEST_F(SpMMTest, opt_performance)
{
    SpMMOpt *spmmer = new SpMMOpt(g, kLen);
    spmmer->preprocess(p_in_feat_vec, p_out_feat_vec);
    auto time = getAverageTimeWithWarmUp([&]()
                                         { spmmer->run(p_in_feat_vec, p_out_feat_vec); });
    dbg(time);
}
