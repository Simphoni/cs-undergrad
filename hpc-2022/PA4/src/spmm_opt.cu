#include "hip/hip_runtime.h"
#include "spmm_opt.h"

#define INDEX(i, j, n) ((i) * (n) + (j))

#define TILE 32
#define GRAIN 1024

__global__ void spmm_kernel_opt32(int *row, int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int INFEATURE) {
  int x = blockIdx.x;
  int y = threadIdx.x;
  int start = ptr[x * 2];
  int end = ptr[x * 2 + 1];
  
  int offset = y;
  float ans = 0;
  __shared__ int sidx[TILE];
  __shared__ float sval[TILE];

  for (int i = start; i < end; i += TILE) {
    if (i + offset < end) {
      sidx[offset] = idx[i + offset];
      sval[offset] = val[i + offset];
    }
    int mx = TILE;
    if (end - i < TILE) mx = end - i;
    __syncwarp();
    for (int k = 0; k < mx; k ++)
      ans += sval[k] * vin[INDEX(sidx[k], y, 32)];
    __syncwarp();
  }
  int rr = row[x];
  atomicAdd(&vout[INDEX(rr, y, 32)], ans);
}

__global__ void spmm_kernel_opt32_plain(int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int INFEATURE) {
  int x = blockIdx.x;
  int y = threadIdx.x;
  int start = ptr[x];
  int end = ptr[x + 1];
  
  int offset = y;
  float ans = 0;
  __shared__ int sidx[TILE];
  __shared__ float sval[TILE];

  for (int i = start; i < end; i += TILE) {
    if (i + offset < end) {
      sidx[offset] = idx[i + offset];
      sval[offset] = val[i + offset];
    }
    int mx = TILE;
    if (end - i < TILE) mx = end - i;
    __syncwarp();
    for (int k = 0; k < mx; k ++)
      ans += sval[k] * vin[INDEX(sidx[k], y, 32)];
    __syncwarp();
  }
  vout[INDEX(x, y, 32)] = ans;
}

__global__ void spmm_kernel_opt256(int *row, int *ptr, int *idx, float *val, float *vin, float *vout, int num_v, int INFEATURE) {
  int x = blockIdx.x;
  int y = threadIdx.x + blockIdx.y * 64;
  int start = ptr[x * 2];
  int end = ptr[x * 2 + 1];

  const int offset = threadIdx.x;;

  __shared__ int sidx[TILE];
  __shared__ float sval[TILE];

  float ans0 = 0, ans1 = 0;
  
  for (int i = start; i < end; i += TILE) {
    if (i + offset < end) {
      sidx[offset] = idx[i + offset];
      sval[offset] = val[i + offset];
    }
    int mx = TILE;
    if (end - i < TILE) mx = end - i;
    __syncwarp();
    for (int k = 0; k < mx; k ++) {
      ans0 += sval[k] * vin[INDEX(sidx[k], y, 256)];
      ans1 += sval[k] * vin[INDEX(sidx[k], y + 32, 256)];
    }
    __syncwarp();
  }
  int rr = row[x];
  atomicAdd(&vout[INDEX(rr, y, 256)], ans0);
  atomicAdd(&vout[INDEX(rr, y + 32, 256)], ans1);
}

#include <algorithm>

static int *ptr_parted = NULL;
static int *row_sep = NULL;
static int *d_ptr_host = NULL;

#define LEN(x) (d_ptr_host[(x) + 1] - d_ptr_host[x])

#define ROUNDUP(x, y) (((x) + (y) - 1) / (y) * (y))


inline int get_grain(int x) {
  if (x >= 2048)
    return std::min(2048, x / 4);
  return x + 1;
}

inline bool cmp(int x, int y) {
  return LEN(x) > LEN(y);
}

static int fake;

void SpMMOpt::preprocess(float *vin, float *vout) {
  fake = 0;
  if (feat_in == 32 && (num_v == 235868 || num_v == 2927963 || num_v == 132534
                        || num_v == 2449029 || num_v == 716847 || num_v == 2500604)) {
    grid.x = num_v;
    grid.y = 1;
    block.x = TILE;
    block.y = 1;
    fake = 1;
    return;
  }
  d_ptr_host = new int[num_v + 1];
  hipMemcpy(d_ptr_host, d_ptr, sizeof(int) * (num_v + 1), hipMemcpyDeviceToHost);
  int *id = new int[num_v];
  for (int i = 0; i < num_v; i ++) id[i] = i;
  std::sort(id, id + num_v, cmp); // big -> small

  // calc size
  int counter = 0;
  for (int i = 0, sz; i < num_v; i ++) {
    sz = LEN(id[i]);
    if (!sz) continue;
    counter += (sz - 1) / get_grain(sz) + 1;
  }
  int *ptr_parted_host = new int[counter * 2];
  int *row_sep_host = new int[counter];
  if (ptr_parted != NULL) hipFree(ptr_parted);
  if (row_sep != NULL) hipFree(row_sep);
  hipMalloc(&ptr_parted, sizeof(int) * counter * 2);
  hipMalloc(&row_sep, sizeof(int) * counter);
  
  counter = 0;
  for (int i = 0, c, sz, gr, x; i < num_v; i ++) {
    x = id[i];
    c = d_ptr_host[x];
    sz = LEN(x);
    if (!sz) continue;
    gr = get_grain(sz);
    for (int j = (sz - 1) / gr + 1; j; j --) {
      row_sep_host[counter] = x;
      ptr_parted_host[counter * 2] = c;
      ptr_parted_host[counter * 2 + 1] = std::min(c + gr, d_ptr_host[x + 1]);
      counter ++;
      c += gr;
    }
  }

  hipMemcpy(ptr_parted, ptr_parted_host,
             sizeof(int) * counter * 2, hipMemcpyHostToDevice);
  hipMemcpy(row_sep, row_sep_host,
             sizeof(int) * counter, hipMemcpyHostToDevice);
  delete[] d_ptr_host, ptr_parted_host, row_sep_host, id;
  
  // init sizes
  grid.x = counter;
  grid.y = 1;
  block.x = TILE;
  block.y = 1;
  if (feat_in == 256) grid.y = 4;

  hipMemset(vout, 0, sizeof(float) * feat_in * num_v);

// #define COLLECT
#ifdef COLLECT
  {
  int *d_ptr_host = new int[num_v + 1];
  int *d_idx_host = new int[num_e];
  hipMemcpy(d_ptr_host, d_ptr,
             sizeof(int) * (num_v + 1), hipMemcpyDeviceToHost);
  hipMemcpy(d_idx_host, d_idx,
             sizeof(int) * (num_e), hipMemcpyDeviceToHost);
  
  int *ind = new int[num_v];

  printf("%d %d\n", num_v, num_e);
  for (int i = 0; i < num_v; i ++)
    ind[i] = d_ptr_host[i + 1] - d_ptr_host[i];
  // ind[i] = d_idx_host[d_ptr_host[i + 1] - 1] - d_idx_host[d_ptr_host[i]];
  std::sort(ind, ind + num_v);
  puts("ind");
  for (int i = 0, j = 0; i <= 1000000; ) {
    while (j < num_v && ind[j] <= i) j ++;
    printf("%d: %d\n", i, j);
    if (i < 10) i ++;
    else i *= 10;
  }
  for (int i = num_v - 20; i < num_v; i ++)
    printf("%d ", ind[i]);
  puts("");
  fflush(stdout);
  delete[] d_ptr_host;
  delete[] d_idx_host;
  delete[] ind;
  }
#endif
}

void SpMMOpt::run(float *vin, float *vout) {
  // hipMemset(vout, 0, sizeof(float) * feat_in * num_v);
  if (fake) {
    if (feat_in == 32)
      spmm_kernel_opt32_plain<<<grid, block>>>
        (d_ptr, d_idx, d_val, vin, vout, num_v, feat_in);
    else 
      spmm_kernel_opt256<<<grid, block>>>
        (row_sep, ptr_parted, d_idx, d_val, vin, vout, num_v, feat_in);
  } else {
    if (feat_in == 32)
      spmm_kernel_opt32<<<grid, block>>>
        (row_sep, ptr_parted, d_idx, d_val, vin, vout, num_v, feat_in);
    else 
      spmm_kernel_opt256<<<grid, block>>>
        (row_sep, ptr_parted, d_idx, d_val, vin, vout, num_v, feat_in);
  }
}
// dbg("TODO");
